#include "hip/hip_runtime.h"

/*
    pbrt source code is Copyright(c) 1998-2016
                        Matt Pharr, Greg Humphreys, and Wenzel Jakob.

    This file is part of pbrt.

    Redistribution and use in source and binary forms, with or without
    modification, are permitted provided that the following conditions are
    met:

    - Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.

    - Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.

    THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
    IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
    TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
    PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
    HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
    SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
    LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
    DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
    THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
    (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
    OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

 */


// core/shape.cpp*
#include "shape.cuh"
#include "stats.cuh"
#include "lowdiscrepancy.cuh"
#include "sampling.cuh"
#include "paramset.cuh"
#include "efloat.cuh"

namespace pbrt {

// Shape Method Definitions
Shape::~Shape() {}

STAT_COUNTER("Scene/Shapes created", nShapesCreated);
// Shape::Shape(const Transform *ObjectToWorld, const Transform *WorldToObject,
//              bool reverseOrientation)
//     : ObjectToWorld(ObjectToWorld),
//       WorldToObject(WorldToObject),
//       reverseOrientation(reverseOrientation),
//       transformSwapsHandedness(ObjectToWorld->SwapsHandedness()) {
//     ++nShapesCreated;
// }

Shape::Shape(const Transform *ObjectToWorld, const Transform *WorldToObject,
           bool reverseOrientation, Float radius, Float zMin, Float zMax,
           Float phiMax)
        : ObjectToWorld(ObjectToWorld),
          WorldToObject(WorldToObject),
          reverseOrientation(reverseOrientation),
          transformSwapsHandedness(ObjectToWorld->SwapsHandedness()),
          radius(radius),
          zMin(Clamp(std::min(zMin, zMax), -radius, radius)),
          zMax(Clamp(std::max(zMin, zMax), -radius, radius)),
          thetaMin(std::acos(Clamp(std::min(zMin, zMax) / radius, -1, 1))),
          thetaMax(std::acos(Clamp(std::max(zMin, zMax) / radius, -1, 1))),
          phiMax(Radians(Clamp(phiMax, 0, 360))) {
              ++nShapesCreated;
          }

// __both__
// Bounds3f Shape::WorldBound() const { return (*ObjectToWorld)(ObjectBound()); }
// __both__
// Interaction Shape::Sample(const Interaction &ref, const Point2f &u,
//                           Float *pdf) const {
//     Interaction intr = Sample(u, pdf);
//     Vector3f wi = intr.p - ref.p;
//     if (wi.LengthSquared() == 0)
//         *pdf = 0;
//     else {
//         wi = Normalize(wi);
//         // Convert from area measure, as returned by the Sample() call
//         // above, to solid angle measure.
//         *pdf *= DistanceSquared(ref.p, intr.p) / AbsDot(intr.n, -wi);
//         if (isinf(*pdf)) *pdf = 0.f;
//     }
//     return intr;
// }
// __both__
// Float Shape::Pdf(const Interaction &ref, const Vector3f &wi) const {
//     // Intersect sample ray with area light geometry
//     Ray ray = ref.SpawnRay(wi);
//     Float tHit;
//     SurfaceInteraction isectLight;
//     // Ignore any alpha textures used for trimming the shape when performing
//     // this intersection. Hack for the "San Miguel" scene, where this is used
//     // to make an invisible area light.
//     if (!Intersect(ray, &tHit, &isectLight, false)) return 0;

//     // Convert light sample weight to solid angle measure
//     Float pdf = DistanceSquared(ref.p, isectLight.p) /
//                 (AbsDot(isectLight.n, -wi) * Area());
//     if (isinf(pdf)) pdf = 0.f;
//     return pdf;
// }

// Float Shape::SolidAngle(const Point3f &p, int nSamples) const {
//     Interaction ref(p, Normal3f(), Vector3f(), Vector3f(0, 0, 1), 0,
//                     MediumInterface{});
//     double solidAngle = 0;
//     for (int i = 0; i < nSamples; ++i) {
//         Point2f u{RadicalInverse(0, i), RadicalInverse(1, i)};
//         Float pdf;
//         Interaction pShape = Sample(ref, u, &pdf);
//         if (pdf > 0 && !IntersectP(Ray(p, pShape.p - p, .999f))) {
//             solidAngle += 1 / pdf;
//         }
//     }
//     return solidAngle / nSamples;
// }

__both__
Bounds3f Shape::ObjectBound() const {
    return Bounds3f(Point3f(-radius, -radius, zMin),
                    Point3f(radius, radius, zMax));
}
__both__
bool Shape::Intersect(const Ray &r, Float *tHit, SurfaceInteraction *isect) const {
    // ProfilePhase p(Prof::ShapeIntersect);
    Float phi;
    Point3f pHit;
    // Transform _Ray_ to object space
    Vector3f oErr, dErr;
    Ray ray = (*WorldToObject)(r, &oErr, &dErr);

    // Compute quadratic sphere coefficients

    // Initialize _EFloat_ ray coordinate values
    EFloat ox(ray.o.x, oErr.x), oy(ray.o.y, oErr.y), oz(ray.o.z, oErr.z);
    EFloat dx(ray.d.x, dErr.x), dy(ray.d.y, dErr.y), dz(ray.d.z, dErr.z);
    EFloat a = dx * dx + dy * dy + dz * dz;
    EFloat b = 2 * (dx * ox + dy * oy + dz * oz);
    EFloat c = ox * ox + oy * oy + oz * oz - EFloat(radius) * EFloat(radius);

    // Solve quadratic equation for _t_ values
    EFloat t0, t1;
    if (!Quadratic(a, b, c, &t0, &t1)) return false;

    // Check quadric shape _t0_ and _t1_ for nearest intersection
    if (t0.UpperBound() > ray.tMax || t1.LowerBound() <= 0) return false;
    EFloat tShapeHit = t0;
    if (tShapeHit.LowerBound() <= 0) {
        tShapeHit = t1;
        if (tShapeHit.UpperBound() > ray.tMax) return false;
    }

    // Compute sphere hit position and $\phi$
    pHit = ray((Float)tShapeHit);

    // Refine sphere intersection point
    pHit *= radius / Distance(pHit, Point3f(0, 0, 0));
    if (pHit.x == 0 && pHit.y == 0) pHit.x = 1e-5f * radius;
    phi = std::atan2(pHit.y, pHit.x);
    if (phi < 0) phi += 2 * Pi;

    // Test sphere intersection against clipping parameters
    if ((zMin > -radius && pHit.z < zMin) || (zMax < radius && pHit.z > zMax) ||
        phi > phiMax) {
        if (tShapeHit == t1) return false;
        if (t1.UpperBound() > ray.tMax) return false;
        tShapeHit = t1;
        // Compute sphere hit position and $\phi$
        pHit = ray((Float)tShapeHit);

        // Refine sphere intersection point
        pHit *= radius / Distance(pHit, Point3f(0, 0, 0));
        if (pHit.x == 0 && pHit.y == 0) pHit.x = 1e-5f * radius;
        phi = std::atan2(pHit.y, pHit.x);
        if (phi < 0) phi += 2 * Pi;
        if ((zMin > -radius && pHit.z < zMin) ||
            (zMax < radius && pHit.z > zMax) || phi > phiMax)
            return false;
    }

    // Find parametric representation of sphere hit
    Float u = phi / phiMax;
    Float theta = std::acos(Clamp(pHit.z / radius, -1, 1));
    Float v = (theta - thetaMin) / (thetaMax - thetaMin);

    // Compute sphere $\dpdu$ and $\dpdv$
    Float zRadius = pbrt::math::sqrt(pHit.x * pHit.x + pHit.y * pHit.y);
    Float invZRadius = 1 / zRadius;
    Float cosPhi = pHit.x * invZRadius;
    Float sinPhi = pHit.y * invZRadius;
    Vector3f dpdu(-phiMax * pHit.y, phiMax * pHit.x, 0);
    Vector3f dpdv =
        (thetaMax - thetaMin) *
        Vector3f(pHit.z * cosPhi, pHit.z * sinPhi, -radius * std::sin(theta));

    // Compute sphere $\dndu$ and $\dndv$
    Vector3f d2Pduu = -phiMax * phiMax * Vector3f(pHit.x, pHit.y, 0);
    Vector3f d2Pduv =
        (thetaMax - thetaMin) * pHit.z * phiMax * Vector3f(-sinPhi, cosPhi, 0.);
    Vector3f d2Pdvv = -(thetaMax - thetaMin) * (thetaMax - thetaMin) *
                      Vector3f(pHit.x, pHit.y, pHit.z);

    // Compute coefficients for fundamental forms
    Float E = Dot(dpdu, dpdu);
    Float F = Dot(dpdu, dpdv);
    Float G = Dot(dpdv, dpdv);
    Vector3f N = Normalize(Cross(dpdu, dpdv));
    Float e = Dot(N, d2Pduu);
    Float f = Dot(N, d2Pduv);
    Float g = Dot(N, d2Pdvv);

    // Compute $\dndu$ and $\dndv$ from fundamental form coefficients
    Float invEGF2 = 1 / (E * G - F * F);
    Normal3f dndu = Normal3f((f * F - e * G) * invEGF2 * dpdu +
                             (e * F - f * E) * invEGF2 * dpdv);
    Normal3f dndv = Normal3f((g * F - f * G) * invEGF2 * dpdu +
                             (f * F - g * E) * invEGF2 * dpdv);

    // Compute error bounds for sphere intersection
    Vector3f pError = gamma(5) * Abs((Vector3f)pHit);

    // Initialize _SurfaceInteraction_ from parametric information
    *isect = (*ObjectToWorld)(SurfaceInteraction(pHit, pError, Point2f(u, v),
                                                 -ray.d, dpdu, dpdv, dndu, dndv,
                                                 ray.time, this));

    // Update _tHit_ for quadric intersection
    *tHit = (Float)tShapeHit;
    return true;
}
__both__
bool Shape::IntersectP(const Ray &r) const {
    // ProfilePhase p(Prof::ShapeIntersectP);
    Float phi;
    Point3f pHit;
    // Transform _Ray_ to object space
    Vector3f oErr, dErr;
    Ray ray = (*WorldToObject)(r, &oErr, &dErr);

    // Compute quadratic sphere coefficients

    // Initialize _EFloat_ ray coordinate values
    EFloat ox(ray.o.x, oErr.x), oy(ray.o.y, oErr.y), oz(ray.o.z, oErr.z);
    EFloat dx(ray.d.x, dErr.x), dy(ray.d.y, dErr.y), dz(ray.d.z, dErr.z);
    EFloat a = dx * dx + dy * dy + dz * dz;
    EFloat b = 2 * (dx * ox + dy * oy + dz * oz);
    EFloat c = ox * ox + oy * oy + oz * oz - EFloat(radius) * EFloat(radius);

    // Solve quadratic equation for _t_ values
    EFloat t0, t1;
    if (!Quadratic(a, b, c, &t0, &t1)) return false;

    // Check quadric shape _t0_ and _t1_ for nearest intersection
    if (t0.UpperBound() > ray.tMax || t1.LowerBound() <= 0) return false;
    EFloat tShapeHit = t0;
    if (tShapeHit.LowerBound() <= 0) {
        tShapeHit = t1;
        if (tShapeHit.UpperBound() > ray.tMax) return false;
    }

    // Compute sphere hit position and $\phi$
    pHit = ray((Float)tShapeHit);

    // Refine sphere intersection point
    pHit *= radius / Distance(pHit, Point3f(0, 0, 0));
    if (pHit.x == 0 && pHit.y == 0) pHit.x = 1e-5f * radius;
    phi = std::atan2(pHit.y, pHit.x);
    if (phi < 0) phi += 2 * Pi;

    // Test sphere intersection against clipping parameters
    if ((zMin > -radius && pHit.z < zMin) || (zMax < radius && pHit.z > zMax) ||
        phi > phiMax) {
        if (tShapeHit == t1) return false;
        if (t1.UpperBound() > ray.tMax) return false;
        tShapeHit = t1;
        // Compute sphere hit position and $\phi$
        pHit = ray((Float)tShapeHit);

        // Refine sphere intersection point
        pHit *= radius / Distance(pHit, Point3f(0, 0, 0));
        if (pHit.x == 0 && pHit.y == 0) pHit.x = 1e-5f * radius;
        phi = std::atan2(pHit.y, pHit.x);
        if (phi < 0) phi += 2 * Pi;
        if ((zMin > -radius && pHit.z < zMin) ||
            (zMax < radius && pHit.z > zMax) || phi > phiMax)
            return false;
    }
    return true;
}
__both__
Float Shape::Area() const { return phiMax * radius * (zMax - zMin); }
__both__
Interaction Shape::Sample(const Point2f &u, Float *pdf) const {
    Point3f pObj = Point3f(0, 0, 0) + radius * UniformSampleSphere(u);
    Interaction it;
    it.n = Normalize((*ObjectToWorld)(Normal3f(pObj.x, pObj.y, pObj.z)));
    if (reverseOrientation) it.n *= -1;
    // Reproject _pObj_ to sphere surface and compute _pObjError_
    pObj *= radius / Distance(pObj, Point3f(0, 0, 0));
    Vector3f pObjError = gamma(5) * Abs((Vector3f)pObj);
    it.p = (*ObjectToWorld)(pObj, pObjError, &it.pError);
    *pdf = 1 / Area();
    return it;
}
__both__
Interaction Shape::Sample(const Interaction &ref, const Point2f &u,
                           Float *pdf) const {
    Point3f pCenter = (*ObjectToWorld)(Point3f(0, 0, 0));

    // Sample uniformly on sphere if $\pt{}$ is inside it
    Point3f pOrigin =
        OffsetRayOrigin(ref.p, ref.pError, ref.n, pCenter - ref.p);
    if (DistanceSquared(pOrigin, pCenter) <= radius * radius) {
        Interaction intr = Sample(u, pdf);
        Vector3f wi = intr.p - ref.p;
        if (wi.LengthSquared() == 0)
            *pdf = 0;
        else {
            // Convert from area measure returned by Sample() call above to
            // solid angle measure.
            wi = Normalize(wi);
            *pdf *= DistanceSquared(ref.p, intr.p) / AbsDot(intr.n, -wi);
        }
        if (isinf(*pdf)) *pdf = 0.f;
        return intr;
    }

    // Sample sphere uniformly inside subtended cone

    // Compute coordinate system for sphere sampling
    Float dc = Distance(ref.p, pCenter);
    Float invDc = 1 / dc;
    Vector3f wc = (pCenter - ref.p) * invDc;
    Vector3f wcX, wcY;
    CoordinateSystem(wc, &wcX, &wcY);

    // Compute $\theta$ and $\phi$ values for sample in cone
    Float sinThetaMax = radius * invDc;
    Float sinThetaMax2 = sinThetaMax * sinThetaMax;
    Float invSinThetaMax = 1 / sinThetaMax;
    Float cosThetaMax = pbrt::math::sqrt(max((Float)0.f, 1 - sinThetaMax2));

    Float cosTheta  = (cosThetaMax - 1) * u[0] + 1;
    Float sinTheta2 = 1 - cosTheta * cosTheta;

    if (sinThetaMax2 < 0.00068523f /* sin^2(1.5 deg) */) {
        /* Fall back to a Taylor series expansion for small angles, where
           the standard approach suffers from severe cancellation errors */
        sinTheta2 = sinThetaMax2 * u[0];
        cosTheta = pbrt::math::sqrt(1 - sinTheta2);
    }

    // Compute angle $\alpha$ from center of sphere to sampled point on surface
    Float cosAlpha = sinTheta2 * invSinThetaMax +
        cosTheta * pbrt::math::sqrt(max((Float)0.f, 1.f - sinTheta2 * invSinThetaMax * invSinThetaMax));
    Float sinAlpha = pbrt::math::sqrt(max((Float)0.f, 1.f - cosAlpha*cosAlpha));
    Float phi = u[1] * 2 * Pi;

    // Compute surface normal and sampled point on sphere
    Vector3f nWorld =
        SphericalDirection(sinAlpha, cosAlpha, phi, -wcX, -wcY, -wc);
    Point3f pWorld = pCenter + radius * Point3f(nWorld.x, nWorld.y, nWorld.z);

    // Return _Interaction_ for sampled point on sphere
    Interaction it;
    it.p = pWorld;
    it.pError = gamma(5) * Abs((Vector3f)pWorld);
    it.n = Normal3f(nWorld);
    if (reverseOrientation) it.n *= -1;

    // Uniform cone PDF.
    *pdf = 1 / (2 * Pi * (1 - cosThetaMax));

    return it;
}
__both__
Float Shape::Pdf(const Interaction &ref, const Vector3f &wi) const {
    Point3f pCenter = (*ObjectToWorld)(Point3f(0, 0, 0));
    // Return uniform PDF if point is inside sphere
    Point3f pOrigin =
        OffsetRayOrigin(ref.p, ref.pError, ref.n, pCenter - ref.p);
    if (DistanceSquared(pOrigin, pCenter) <= radius * radius){
        // Intersect sample ray with area light geometry
        Ray ray = ref.SpawnRay(wi);
        Float tHit;
        SurfaceInteraction isectLight;
        // Ignore any alpha textures used for trimming the shape when performing
        // this intersection. Hack for the "San Miguel" scene, where this is used
        // to make an invisible area light.
        if (!Intersect(ray, &tHit, &isectLight)) return 0;

        // Convert light sample weight to solid angle measure
        Float pdf = DistanceSquared(ref.p, isectLight.p) /
                    (AbsDot(isectLight.n, -wi) * Area());
        if (isinf(pdf)) pdf = 0.f;
        return pdf;
    }

    // Compute general sphere PDF
    Float sinThetaMax2 = radius * radius / DistanceSquared(ref.p, pCenter);
    Float cosThetaMax = pbrt::math::sqrt(max((Float)0, 1 - sinThetaMax2));
    return UniformConePdf(cosThetaMax);
}

Float Shape::SolidAngle(const Point3f &p, int nSamples) const {
    Point3f pCenter = (*ObjectToWorld)(Point3f(0, 0, 0));
    if (DistanceSquared(p, pCenter) <= radius * radius)
        return 4 * Pi;
    Float sinTheta2 = radius * radius / DistanceSquared(p, pCenter);
    Float cosTheta = pbrt::math::sqrt(std::max((Float)0, 1 - sinTheta2));
    return (2 * Pi * (1 - cosTheta));
}

}  // namespace pbrt
