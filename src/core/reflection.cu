#include "hip/hip_runtime.h"

/*
    pbrt source code is Copyright(c) 1998-2016
                        Matt Pharr, Greg Humphreys, and Wenzel Jakob.

    This file is part of pbrt.

    Redistribution and use in source and binary forms, with or without
    modification, are permitted provided that the following conditions are
    met:

    - Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.

    - Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.

    THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
    IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
    TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
    PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
    HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
    SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
    LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
    DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
    THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
    (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
    OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

 */

// core/reflection.cpp*
#include "reflection.cuh"
#include "spectrum.cuh"
#include "sampler.cuh"
#include "sampling.cuh"
#include "interpolation.cuh"
#include "scene.cuh"
#include "interaction.cuh"
#include "stats.cuh"
#include <stdarg.h>

namespace pbrt {

// BxDF Utility Functions
Float FrDielectric(Float cosThetaI, Float etaI, Float etaT) {
    cosThetaI = Clamp(cosThetaI, -1, 1);
    // Potentially swap indices of refraction
    bool entering = cosThetaI > 0.f;
    if (!entering) {
        std::swap(etaI, etaT);
        cosThetaI = std::abs(cosThetaI);
    }

    // Compute _cosThetaT_ using Snell's law
    Float sinThetaI = pbrt::math::sqrt(std::max((Float)0, 1 - cosThetaI * cosThetaI));
    Float sinThetaT = etaI / etaT * sinThetaI;

    // Handle total internal reflection
    if (sinThetaT >= 1) return 1;
    Float cosThetaT = pbrt::math::sqrt(std::max((Float)0, 1 - sinThetaT * sinThetaT));
    Float Rparl = ((etaT * cosThetaI) - (etaI * cosThetaT)) /
                  ((etaT * cosThetaI) + (etaI * cosThetaT));
    Float Rperp = ((etaI * cosThetaI) - (etaT * cosThetaT)) /
                  ((etaI * cosThetaI) + (etaT * cosThetaT));
    return (Rparl * Rparl + Rperp * Rperp) / 2;
}

// https://seblagarde.wordpress.com/2013/04/29/memo-on-fresnel-equations/
Spectrum FrConductor(Float cosThetaI, const Spectrum &etai,
                     const Spectrum &etat, const Spectrum &k) {
    cosThetaI = Clamp(cosThetaI, -1, 1);
    Spectrum eta = etat / etai;
    Spectrum etak = k / etai;

    Float cosThetaI2 = cosThetaI * cosThetaI;
    Float sinThetaI2 = 1. - cosThetaI2;
    Spectrum eta2 = eta * eta;
    Spectrum etak2 = etak * etak;

    Spectrum t0 = eta2 - etak2 - sinThetaI2;
    Spectrum a2plusb2 = Sqrt(t0 * t0 + 4 * eta2 * etak2);
    Spectrum t1 = a2plusb2 + cosThetaI2;
    Spectrum a = Sqrt(0.5f * (a2plusb2 + t0));
    Spectrum t2 = (Float)2 * cosThetaI * a;
    Spectrum Rs = (t1 - t2) / (t1 + t2);

    Spectrum t3 = cosThetaI2 * a2plusb2 + sinThetaI2 * sinThetaI2;
    Spectrum t4 = t2 * sinThetaI2;
    Spectrum Rp = Rs * (t3 - t4) / (t3 + t4);

    return 0.5 * (Rp + Rs);
}

// // BxDF Method Definitions
// Spectrum LambertianReflection::f(const Vector3f &wo, const Vector3f &wi) const {
//     return R * InvPi;
// }

std::string BxDF::ToString() const {
    return std::string("[ LambertianReflection R: ") + R.ToString() +
           std::string(" ]");
}

// Spectrum BxDF::Sample_f(const Vector3f &wo, Vector3f *wi, const Point2f &u,
//                         Float *pdf, BxDFType *sampledType) const {
//     // Cosine-sample the hemisphere, flipping the direction if necessary
//     *wi = CosineSampleHemisphere(u);
//     if (wo.z < 0) wi->z *= -1;
//     *pdf = Pdf(wo, *wi);
//     return f(wo, *wi);
// }

// Float BxDF::Pdf(const Vector3f &wo, const Vector3f &wi) const {
//     return SameHemisphere(wo, wi) ? AbsCosTheta(wi) * InvPi : 0;
// }

// Spectrum BxDF::rho(const Vector3f &w, int nSamples, const Point2f *u) const {
//     Spectrum r(0.);
//     for (int i = 0; i < nSamples; ++i) {
//         // Estimate one term of $\rho_\roman{hd}$
//         Vector3f wi;
//         Float pdf = 0;
//         Spectrum f = Sample_f(w, &wi, u[i], &pdf);
//         if (pdf > 0) r += f * AbsCosTheta(wi) / pdf;
//     }
//     return r / nSamples;
// }

// Spectrum BxDF::rho(int nSamples, const Point2f *u1, const Point2f *u2) const {
//     Spectrum r(0.f);
//     for (int i = 0; i < nSamples; ++i) {
//         // Estimate one term of $\rho_\roman{hh}$
//         Vector3f wo, wi;
//         wo = UniformSampleHemisphere(u1[i]);
//         Float pdfo = UniformHemispherePdf(), pdfi = 0;
//         Spectrum f = Sample_f(wo, &wi, u2[i], &pdfi);
//         if (pdfi > 0)
//             r += f * AbsCosTheta(wi) * AbsCosTheta(wo) / (pdfo * pdfi);
//     }
//     return r / (Pi * nSamples);
// }

// BSDF Method Definitions
// Spectrum BSDF::f(const Vector3f &woW, const Vector3f &wiW,
//                  BxDFType flags) const {
//     // ProfilePhase pp(Prof::BSDFEvaluation);
//     Vector3f wi = WorldToLocal(wiW), wo = WorldToLocal(woW);
//     if (wo.z == 0) return 0.;
//     bool reflect = Dot(wiW, ng) * Dot(woW, ng) > 0;
//     Spectrum f(0.f);
//     for (int i = 0; i < nBxDFs; ++i)
//         if (bxdfs[i]->MatchesFlags(flags) &&
//             ((reflect && (bxdfs[i]->type & BSDF_REFLECTION)) ||
//              (!reflect && (bxdfs[i]->type & BSDF_TRANSMISSION))))
//             f += bxdfs[i]->f(wo, wi);
//     return f;
// }

Spectrum BSDF::rho(int nSamples, const Point2f *samples1,
                   const Point2f *samples2, BxDFType flags) const {
    Spectrum ret(0.f);
    for (int i = 0; i < nBxDFs; ++i)
        if (bxdfs[i]->MatchesFlags(flags))
            ret += bxdfs[i]->rho(nSamples, samples1, samples2);
    return ret;
}

Spectrum BSDF::rho(const Vector3f &woWorld, int nSamples, const Point2f *samples,
                   BxDFType flags) const {
    Vector3f wo = WorldToLocal(woWorld);
    Spectrum ret(0.f);
    for (int i = 0; i < nBxDFs; ++i)
        if (bxdfs[i]->MatchesFlags(flags))
            ret += bxdfs[i]->rho(wo, nSamples, samples);
    return ret;
}

// Spectrum BSDF::Sample_f(const Vector3f &woWorld, Vector3f *wiWorld,
//                         const Point2f &u, Float *pdf, BxDFType type,
//                         BxDFType *sampledType) const {
//     // ProfilePhase pp(Prof::BSDFSampling);
//     // Choose which _BxDF_ to sample
//     int matchingComps = NumComponents(type);
//     if (matchingComps == 0) {
//         *pdf = 0;
//         if (sampledType) *sampledType = BxDFType(0);
//         return Spectrum(0);
//     }
//     int comp =
//         min((int)std::floor(u[0] * matchingComps), matchingComps - 1);

//     // Get _BxDF_ pointer for chosen component
//     BxDF *bxdf = nullptr;
//     int count = comp;
//     for (int i = 0; i < nBxDFs; ++i)
//         if (bxdfs[i]->MatchesFlags(type) && count-- == 0) {
//             bxdf = bxdfs[i];
//             break;
//         }
//     // CHECK(bxdf != nullptr);
//     // VLOG(2) << "BSDF::Sample_f chose comp = " << comp << " / matching = " <<
//         // matchingComps << ", bxdf: " << bxdf->ToString();

//     // Remap _BxDF_ sample _u_ to $[0,1)^2$
//     Point2f uRemapped(min(u[0] * matchingComps - comp, OneMinusEpsilon),
//                       u[1]);

//     // Sample chosen _BxDF_
//     Vector3f wi, wo = WorldToLocal(woWorld);
//     if (wo.z == 0) return 0.;
//     *pdf = 0;
//     if (sampledType) *sampledType = bxdf->type;
//     Spectrum f = bxdf->Sample_f(wo, &wi, uRemapped, pdf, sampledType);
//     // VLOG(2) << "For wo = " << wo << ", sampled f = " << f << ", pdf = "
//     //         << *pdf << ", ratio = " << ((*pdf > 0) ? (f / *pdf) : Spectrum(0.))
//     //         << ", wi = " << wi;
//     if (*pdf == 0) {
//         if (sampledType) *sampledType = BxDFType(0);
//         return 0;
//     }
//     *wiWorld = LocalToWorld(wi);

//     // Compute overall PDF with all matching _BxDF_s
//     if (!(bxdf->type & BSDF_SPECULAR) && matchingComps > 1)
//         for (int i = 0; i < nBxDFs; ++i)
//             if (bxdfs[i] != bxdf && bxdfs[i]->MatchesFlags(type))
//                 *pdf += bxdfs[i]->Pdf(wo, wi);
//     if (matchingComps > 1) *pdf /= matchingComps;

//     // Compute value of BSDF for sampled direction
//     if (!(bxdf->type & BSDF_SPECULAR)) {
//         bool reflect = Dot(*wiWorld, ng) * Dot(woWorld, ng) > 0;
//         f = 0.;
//         for (int i = 0; i < nBxDFs; ++i)
//             if (bxdfs[i]->MatchesFlags(type) &&
//                 ((reflect && (bxdfs[i]->type & BSDF_REFLECTION)) ||
//                  (!reflect && (bxdfs[i]->type & BSDF_TRANSMISSION))))
//                 f += bxdfs[i]->f(wo, wi);
//     }
//     // VLOG(2) << "Overall f = " << f << ", pdf = " << *pdf << ", ratio = "
//     //         << ((*pdf > 0) ? (f / *pdf) : Spectrum(0.));
//     return f;
// }

// Float BSDF::Pdf(const Vector3f &woWorld, const Vector3f &wiWorld,
//                 BxDFType flags) const {
//     // ProfilePhase pp(Prof::BSDFPdf);
//     if (nBxDFs == 0.f) return 0.f;
//     Vector3f wo = WorldToLocal(woWorld), wi = WorldToLocal(wiWorld);
//     if (wo.z == 0) return 0.;
//     Float pdf = 0.f;
//     int matchingComps = 0;
//     for (int i = 0; i < nBxDFs; ++i)
//         if (bxdfs[i]->MatchesFlags(flags)) {
//             ++matchingComps;
//             pdf += bxdfs[i]->Pdf(wo, wi);
//         }
//     Float v = matchingComps > 0 ? pdf / matchingComps : 0.f;
//     return v;
// }

std::string BSDF::ToString() const {
    std::string s = StringPrintf("[ BSDF eta: %f nBxDFs: %d", eta, nBxDFs);
    for (int i = 0; i < nBxDFs; ++i)
        s += StringPrintf("\n  bxdfs[%d]: ", i) + bxdfs[i]->ToString();
    return s + std::string(" ]");
}

}  // namespace pbrt
